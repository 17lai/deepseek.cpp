#include "hip/hip_runtime.h"
#include "model.h"

#include <cfloat>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define FULL_MASK 0xffffffff

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice));
	return device;
}

[[maybe_unused]] static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

[[maybe_unused]] static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void* upload_cuda(void* host, size_t size) {
	return cuda_devicecopy(host, size);
}

extern "C" void register_cuda_host(void* host, size_t size) {
  CUDA_CHECK(hipHostRegister(host, size, hipHostRegisterDefault));
}

extern "C" void free_cuda(void* device) {
  CUDA_CHECK(hipFree(device));
}

extern "C" void unregister_cuda_host(void* host) {
  CUDA_CHECK(hipHostUnregister(host));
}

static int warp_size = 0;
static int max_threads_per_block = 0;

extern "C" void set_cuda_device(int device) {
  CUDA_CHECK(hipSetDevice(device));
  CUDA_CHECK(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, device));
  CUDA_CHECK(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, device));
}

__device__ 
inline float warp_reduce_sum(float val) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);

  return val;
}

__device__ 
inline float warp_all_reduce_max(float val) {
	// Max reduction across a warp.
	// All threads will contain the max of all threads in the warp.
  for (int mask = warpSize/2; mask > 0; mask /= 2) {
    val = max(val, __shfl_xor_sync(FULL_MASK, val, mask));
  }
  return val;
}

__device__ 
inline float block_all_reduce_max(float val) {
	// Max reduction across a 1-D block implemented as double warp max reduction.
	// All threads will contain the max of all threads in the block.
	
	// Will hold results of all warps.
	// Capacity 32 since there can be at most 32 warps in a thread.
  __shared__ float shared[32];
  const int wid  = threadIdx.x / warpSize;
  const int lane = threadIdx.x % warpSize;

  val = warp_all_reduce_max(val);

  if (blockDim.x < warpSize) return val;
  if (lane == 0) shared[wid] = val;

  __syncthreads();

  if ( wid == 0 ) {
	  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : -FLT_MAX;
  }
  val = warp_all_reduce_max(val);
  if (lane == 0) shared[wid] = val;
  
  __syncthreads();
  
  return shared[0];
}

__device__ 
inline float warp_all_reduce_sum(float val) {
	// Sum reduction across a warp.
	// All threads will contain the sum of all threads in the warp.
  for (int mask = warpSize/2; mask > 0; mask /= 2) {
    val += __shfl_xor_sync(FULL_MASK, val, mask);
  }
  return val;
}

__device__ 
inline float block_all_reduce_sum(float val) {
	// Sum reduction across a 1-D block implemented as double warp sum reduction.
	// All threads will contain the sum of all threads in the block.
	
	// Will hold results of all warps.
	// Capacity 32 since there can be at most 32 warps in a thread.
  __shared__ float shared[32];
  const int wid  = threadIdx.x / warpSize;
  const int lane = threadIdx.x % warpSize;

  val = warp_all_reduce_sum(val);

  if (blockDim.x < warpSize) return val;
  if (lane == 0) shared[wid] = val;

  __syncthreads();

  if ( wid == 0 ) {
	  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0;
  }
  val = warp_all_reduce_sum(val);
  if (lane == 0) shared[wid] = val;
  
  __syncthreads();
  
  return shared[0];
}

__device__
inline float matmul_row(const float* row, const float* x, int offset, int dim) {
	float sum = 0.0;
	for (int j = offset; j < dim; j += warpSize) {
		float v = row[j] * x[j];
		sum += v;
	}
	return warp_reduce_sum(sum);
}

__global__
void matmul(const float* A, const float* x, int n, int d, float* out) {
	// A (d,n) @ x (n,) -> out (d,)
	// PRECOND: Block is 1-D.
	int i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
	if (i >= d) return;
	// Since block is 1-dimensional, thread ID is same as threadIdx.x,
	// and warp partitions thread IDs
	int offset = threadIdx.x % warpSize;
	float rowSum = matmul_row(&A[n * i], x, offset, n);
	if (offset == 0) {
		out[i] = rowSum;
	}
}

__global__
void attn(
	const float* kb, 
	const float* q, 
	int head_dim, 
	int kv_len, 
	int max_seq_len, 
	int n_heads, 
	float* out
) {
	int group = blockIdx.y;
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int h = blockIdx.y * blockDim.y + threadIdx.y;
	if (t >= kv_len || h >= n_heads) return;
	
	const float* query = q + h * head_dim;
	const float* key = kb + max_seq_len * group + head_dim * t;
	float score = 0.0;
	for (int i = 0; i < head_dim; i++) {
		score += query[i] * key[i];
	}
}

__global__
void attn_softmax(
	const float* att, 
	int seq_len, 
	int max_seq_len, 
	int n_heads, 
	float* out
) {
	int offset = threadIdx.x;
	int h = blockIdx.x;
	int block_size = blockDim.x;
	if (h >= n_heads) return;
	
	const float* atth = att + max_seq_len * h;
	float* outh = out + max_seq_len * h;
	
	float score_max = -FLT_MAX;
	for (int t = offset; t < seq_len; t += block_size) {
		if (atth[t] > score_max) {
			score_max = atth[t];
		}
	}
	score_max = block_all_reduce_max(score_max);
	float score_sum = 0.0f;
	for (int t = offset; t < seq_len; t += block_size) {
		outh[t] = expf(atth[t] - score_max);
		score_sum += outh[t];
	}
	score_sum = block_all_reduce_sum(score_sum);
	for (int t = offset; t < seq_len; t += block_size) {
		outh[t] /= score_sum;
	}
}

__global__
void att_mix(
	const float* vb, 
	const float* att, 
	int head_dim, 
	int n_heads, 
	int n_kv_heads,
	int seq_len, 
	int max_seq_len, 
	float* out
) {
	// PRECOND: blocks are 1-D and blockDim.x == warpSize
	int h = blockIdx.x;
	int group_size = n_heads / n_kv_heads;
	int g = h / group_size;
	int i = blockIdx.y;
	int offset = threadIdx.x;
	
	const float* atth = att + max_seq_len * h;
	const float* vh = vb + max_seq_len * head_dim * g;
	float* outh = out + head_dim * h;
	
	float sum = 0.0;
	for (int t = offset; t < seq_len; t += warpSize) {
		sum += vh[head_dim * t + offset] * atth[t];
	}
	sum = warp_reduce_sum(sum);
	if (offset == 0) outh[i] = sum;
}

__global__
void rmsnorm(const float* x, const float* weight, int size, float eps, float* out) {
	// PRECOND: only one 1-D block is launched
	float rms = 0.0;
	int offset = threadIdx.x;
	for (int i = offset; i < size; i += blockDim.x) {
		rms += x[i] * x[i];
	}
	rms = block_all_reduce_sum(rms);
	rms = sqrtf(rms / size + eps);
	float scale = 1.0 / rms;
	for (int i = offset; i < size; i += blockDim.x) {
		out[i] = x[i] * scale * weight[i];
	}
}

__global__
void rope(
	const float* x, int d, int head_dim, int pos, float theta, int rotary_dim, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = 2 * (blockDim.x * blockIdx.x + threadIdx.x);
	if (i >= d) return;
	
	int j_head = i % head_dim;
	float freq = j_head >= rotary_dim ? 0.f : 1.0f / powf(theta, (float)j_head / (float)rotary_dim);
	float val = pos * freq;
	float fcr = cosf(val);
	float fci = sinf(val);
	
	float v0 = x[i];
	float v1 = x[i + 1];
	out[i] = v0 * fcr - v1 * fci;
	out[i + 1] = v0 * fci + v1 * fcr;
}

__global__
void add_residuals(
	const float* x, const float* y, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = x[i] + y[i];
}

__global__
void clip(
	const float* x, float v, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = x[i] < -v ? -v : (x[i] > v ? v : x[i]);
}

__global__
void glu_silu(
	const float* x, const float* weight, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	out[i] = weight[i] * x[i] / (1.0f + expf(-x[i]));
}

__global__
void glu_gelu(
	const float* x, const float* weight, int d, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= d) return;
	
	float v = x[i];
	out[i] =
		weight[i] * 
		0.5f * v * (1.0f + tanhf(0.797885f * (v + 0.044715f * v * v * v)));
}

// TODO: consolidate copy_embedding and copy_kv_entry into 1 memcpy kernel
__global__
void copy_embedding(
	const float* token_embedding_table, int dim, int token, float* out
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= dim) return;
	
	const float* v = token_embedding_table + dim * token;
	out[i] = v[i];
}

__global__
void copy_kv_entry(
	const float* in, int kv_pos, int kv_dim, float* kb
) {
	// PRECOND: grid and blocks are 1-D
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= kv_dim) return;
	
	kb[kv_pos * kv_dim + i] = in[i];
}

void Block::_block_cuda(
	InferenceState& s, int pos, int kv_pos, int kv_len
) const {
	// TODO: support f16
	const Config& c = *_config;
	
	// attention pre-norm
	switch (c.norm_type) {
    case LayerNormType::RMSNorm: {
	    rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_att_weight(), c.dim, c.norm_eps, s.xb()
			);
      break;
    }
  }
  
  int q_dim = c.n_heads * c.head_dim;
  int kv_dim = c.n_kv_heads * c.head_dim;

  // qkv matmuls for this position
  matmul<<<
    (q_dim + warp_size - 1)/warp_size, warp_size
  >>>(wq<float>(), s.xb(), c.dim, q_dim, s.q());
  matmul<<<
    (kv_dim + warp_size - 1)/warp_size, warp_size
  >>>(wk<float>(), s.xb(), c.dim, kv_dim, s.k());
  matmul<<<
    (kv_dim + warp_size - 1)/warp_size, warp_size
  >>>(wv<float>(), s.xb(), c.dim, kv_dim, s.v());
  
  // some models require clipping qkv values
  clip<<<
	  (q_dim + max_threads_per_block - 1)/max_threads_per_block, 
	  max_threads_per_block
  >>>(s.q(), c.qkv_clip, q_dim, s.q());
  clip<<<
	  (kv_dim + max_threads_per_block - 1)/max_threads_per_block, 
	  max_threads_per_block
  >>>(s.k(), c.qkv_clip, kv_dim, s.k());
  clip<<<
	  (kv_dim + max_threads_per_block - 1)/max_threads_per_block,
	  max_threads_per_block
  >>>(s.v(), c.qkv_clip, kv_dim, s.v());
  
  // RoPE relative positional encoding: complex-valued rotate q and k in each head
  rope<<<
	  (q_dim + max_threads_per_block - 1)/max_threads_per_block,
	  max_threads_per_block
  >>>(
		s.q(), q_dim, c.head_dim, pos, c.rope_theta, c.rotary_dim, s.q()
	);
	rope<<<
	  (kv_dim + max_threads_per_block - 1)/max_threads_per_block,
	  max_threads_per_block
  >>>(
		s.k(), kv_dim, c.head_dim, pos, c.rope_theta, c.rotary_dim, s.k()
	);
	
	// key and value point to the kv cache
	float* kb = key_cache();
	float* vb = value_cache();
	copy_kv_entry<<<
		(kv_dim + max_threads_per_block - 1)/max_threads_per_block, 
		max_threads_per_block
	>>>(
		s.k(), kv_pos, kv_dim, kb
	);
	copy_kv_entry<<<
		(kv_dim + max_threads_per_block - 1)/max_threads_per_block, 
		max_threads_per_block
	>>>(
		s.v(), kv_pos, kv_dim, vb
	);
	
	// multihead attention: dot products and softmax
	{
		dim3 tpb;
		tpb.x = warp_size;
		tpb.y = max_threads_per_block / warp_size;
		dim3 blocks;
		blocks.x = (kv_len + tpb.x - 1) / tpb.x;
		blocks.y = (c.n_heads + tpb.y - 1) / tpb.y;
		attn<<<blocks, tpb>>>(
			kb, s.q(), c.head_dim, kv_len, c.max_seq_len, c.n_heads, s.att()
		);
		attn_softmax<<<c.n_heads, warp_size>>>(
			s.att(), kv_len, c.max_seq_len, c.n_heads, s.att()
		);
	}
  // multihead attention: mix values with attention scores
	{
		dim3 tpb;
		tpb.x = warp_size;
		dim3 blocks;
		blocks.x = c.n_heads;
		blocks.y = c.head_dim;
		att_mix<<<blocks, tpb>>>(
			vb, s.att(),
			c.head_dim, c.n_heads, c.n_kv_heads, 
			kv_len, c.max_seq_len, s.xb2()
		);
	}
	// final matmul projection via wo, using `hb` as temp storage
	matmul<<<
    (c.dim + warp_size - 1)/warp_size, warp_size
  >>>(wo<float>(), s.att(), q_dim, c.dim, s.hb());
	
	// attn residual back into x
	add_residuals<<<
		(c.dim + max_threads_per_block - 1)/max_threads_per_block, 
		max_threads_per_block
	>>>(
		s.x(), s.hb(), c.dim, s.x()
	);
	
	// ffn pre-norm
	switch (c.norm_type) {
    case LayerNormType::RMSNorm: {
	    rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_ffn_weight(), c.dim, c.norm_eps, s.xb()
			);
      break;
    }
  }
	
	// mix self.w2(F.silu(self.w1(x)) * self.w3(x))
  // Note this is a feedforward with a GLU, not a simple MLP.
  matmul<<<
    (c.hidden_dim + warp_size - 1)/warp_size, warp_size
  >>>(w1<float>(), s.xb(), c.dim, c.hidden_dim, s.hb());
  matmul<<<
    (c.hidden_dim + warp_size - 1)/warp_size, warp_size
  >>>(w3<float>(), s.xb(), c.dim, c.hidden_dim, s.hb2());
  switch (c.act) {
	  case ActivationType::GELU: {
		  glu_gelu<<<
			  (c.hidden_dim + max_threads_per_block - 1)/max_threads_per_block, 
			  max_threads_per_block
		  >>>(
				s.hb(), s.hb2(), c.hidden_dim, s.hb()
			);
		  break;
	  }
	  case ActivationType::SILU: {
		  glu_silu<<<
			  (c.hidden_dim + max_threads_per_block - 1)/max_threads_per_block, 
			  max_threads_per_block
		  >>>(
				s.hb(), s.hb2(), c.hidden_dim, s.hb()
			);
		  break;
	  }
  }
  
  matmul<<<
    (c.dim + warp_size - 1)/warp_size, warp_size
  >>>(w2<float>(), s.hb(), c.hidden_dim, c.dim, s.xb2());
  
	// ffn residual back into x
	add_residuals<<<
		(c.dim + max_threads_per_block - 1)/max_threads_per_block,
		max_threads_per_block
	>>>(
		s.x(), s.xb2(), c.dim, s.x()
	);
}

void Model::_forward_cuda(InferenceState& s, int token, int pos, InferenceMode mode) {
	const Config& c = *config;
	if (c.weight_dtype != DType::F32) {
		assert(false && "unsupported weight dtype for CUDA");
	}
	
	copy_embedding<<<
    (c.dim + max_threads_per_block - 1)/max_threads_per_block,
		max_threads_per_block
	>>>(
		static_cast<float*>(token_embedding_table), c.dim, token, s.x()
	);
	
	// TODO: attention sinks
	int kv_pos = pos % c.max_seq_len;
	int kv_len = pos >= c.max_seq_len ? c.max_seq_len : pos + 1;
	
	// forward all layers in order
	for (auto b : blocks) {
		b->block(s, pos, kv_pos, kv_len);
	}

  if (mode == InferenceMode::HYDRATE_KV_CACHE) {
    // only hydrate the KV cache and don't compute output logits
	  CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
    return;
  }
	
	// final layer norm
	switch (c.norm_type) {
		case LayerNormType::RMSNorm: {
			rmsnorm<<<1, max_threads_per_block>>>(
				s.x(), rms_final_weight, c.dim, c.norm_eps, s.x()
			);
			break;
		}
	}
	
	// classifier into logits
	switch (c.weight_dtype) {
    case DType::F32: {
	    matmul<<<
		    (c.vocab_size + warp_size - 1)/warp_size, warp_size
	    >>>(static_cast<float*>(wcls), s.x(), c.dim, c.vocab_size, s.logits());
      break;
    }
    default: {
      assert(false && "unsupported weight dtype for CUDA");
    }
  }
	
	CUDA_CHECK(hipDeviceSynchronize()); // After this, s.logits contains logits of output token
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
}