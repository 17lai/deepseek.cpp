#include "model.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice));
	return device;
}

[[maybe_unused]] static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

[[maybe_unused]] static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void* upload_cuda(void* host, size_t size) {
	return cuda_devicecopy(host, size);
}

extern "C" void register_cuda_host(void* host, size_t size) {
  CUDA_CHECK(hipHostRegister(&host, size, hipHostRegisterDefault));
}

extern "C" void free_cuda(void* device) {
  CUDA_CHECK(hipFree(device));
}

extern "C" void unregister_cuda_host(void* host) {
  CUDA_CHECK(hipHostUnregister(host));
}